﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(int *nin, double h);

#define N 1000



__device__ bool ChechPoint(double* k, double *p)
{
    if ((*k) * (*k) + (*p) * (*p) <= 1)
        return true;
    return false;
}

__global__ void add(unsigned int* nin)
{
    int i = threadIdx.x;
    int j = blockIdx.x;

    double k = double(i)/N;
    double p = double(j)/N;

    if (ChechPoint(&k, &p))
        atomicAdd(nin, 1);
}

int main()
{


    unsigned int nin = 0;

    unsigned int* dev_nin;
    //Скопировать в gpu
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&dev_nin, sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }

    cudaStatus = hipMemcpy(dev_nin, &nin, sizeof(unsigned int), hipMemcpyHostToDevice);
 

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyHostToDevice failed!");
        return 1;
    }

    //передать в гпу
    add <<<N, N >>>(dev_nin) ;
    cudaStatus = hipMemcpy(&nin, dev_nin, sizeof(unsigned int), hipMemcpyDeviceToHost);
    //освободить память

    printf("%f\n", double(nin) * 4 / N / N);

    hipFree(dev_nin);
    return 0;

}
